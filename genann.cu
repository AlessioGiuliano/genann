#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>

#include "genann.h"
#include <hip/hip_runtime.h>


__global__ void genann_act_sigmoid_kernel(double* lookup, double f)
{
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    double a = sigmoid_dom_min + f * i;
    if (i < LOOKUP_SIZE)
    {
        if (a < -45.0)
            lookup[i] = 0.;
        else if (a > 45.0)
            lookup[i] = 1.;
        else
            lookup[i] = 1.0 / (1 + std::exp(-a));
    }
}

#define cudaSafeCall(call)                                                                                             \
    do                                                                                                                 \
    {                                                                                                                  \
        hipError_t err = call;                                                                                        \
        if (err != hipSuccess)                                                                                        \
        {                                                                                                              \
            std::cout << "Line " << __LINE__ << ": cuda failure (" << hipGetErrorString(err) << ')' << std::endl;                              \
        }                                                                                                              \
    } while (0)

double genann_init_sigmoid_lookup_cuda(double* lookup) {
        const double f = (sigmoid_dom_max - sigmoid_dom_min) / LOOKUP_SIZE;

        double* d_lookup;
        auto size = LOOKUP_SIZE * sizeof(double);

        cudaSafeCall(hipMalloc((void**) &d_lookup, size));
        cudaSafeCall(hipMemcpy((void*) d_lookup, lookup, size, hipMemcpyHostToDevice));

        const size_t threadsPerBlock = 32;
        const size_t nbBlocks = std::ceil(LOOKUP_SIZE / threadsPerBlock);

        genann_act_sigmoid_kernel<<<nbBlocks, threadsPerBlock>>>(d_lookup, f);
        cudaSafeCall(hipGetLastError());

        cudaSafeCall(hipMemcpy((void*)lookup, d_lookup, size, hipMemcpyDeviceToHost));
        cudaSafeCall(hipFree((void*)d_lookup));

        return LOOKUP_SIZE / (sigmoid_dom_max - sigmoid_dom_min);
}
